#include <iostream>
#include <string>

#include "Fontes/Macros/MacrosSO.h"
#include "Fontes/MonteCarlo.h"

using std::cout;
using std::endl;
using std::string;
using std::to_string;

/*
  Método responsável por apresentar uma tela de ajuda com as opções 
  disponíveis por linha de comando. Estas posições são: 

  "--help -h": mostra a tela de ajuda; 
  "--device -d": especifica a GPU para execução da simulação; 
  "--nmontecarlos -m": especifica a quantidade de simulações Monte Carlo.
  "--saidasubciclo -s": controla a saída bitstring em períodos e subciclos.
*/
void help(string exe) {
  cout << "Uso: " << exe << " <opcoes>\n"
       << "Opcoes:\n"
       << "\t--help, -h\t\tMostra esta ajuda\n"
       << "\t--device, -d ID_DEVICE\tEspecifica a GPU para execucao\n"
       << "\t--nmontecarlos, -m QUANT_MCS\tEspecifica a quantidade de simulacoes MC\n"
       << "\t--saidasubciclo, -s SAIDA_SUB\tControla a saida bitstring em periodos e subciclos\n";
}

/*
  Método inicial do programa. 

  Este método é responsável por:
  - interpretar as opções passadas por linha de comando;
  - alterar a GPU utilizada para execução;
  - excluir pasta de saída se já existente;
  - iniciar a execução das simulações Monte Carlo. 

  Valores padrão:
  - "idDevice": 0. Utilizará a primeira GPU encontrada. A id das GPUs pode 
                   ser vista com o utilitário "deviceQuery" presente no 
                   SDK do CUDA. 
  - "quantMCS": 1. Executará uma simulação Monte Carlo com os arquivos de 
                   entrada presentes na pasta "Entradas/MonteCarlo_0". 
                   Se informado um número maior que 1 serão executadas 
                   simulações utilizando os arquivos presentes nas pastas 
                   "Entradas/MonteCarlo_{1}", em que "{1}" designa o id da 
                   simulação, iniciando em "0" até "quantMCS - 1".
*/
int main(int argc, char **argv) {
  int idDevice = 0, quantMCs = 1, saidaSubciclo = 0;

  // Interpreta os argumentos passados por linha do comando, se existentes. 
  if (argc > 1) {
    for (int i = 1; i < argc; i += 2) {
      string textoOpcao(argv[i]);

      if (textoOpcao == "--help" or textoOpcao == "-h") {
        help(argv[0]);
        return 0;
      }
      if (textoOpcao == "--device" or textoOpcao == "-d") {
        idDevice = atoi(argv[i + 1]);
      } 
      if (textoOpcao == "--nmontecarlos" or textoOpcao == "-m") {
        quantMCs = atoi(argv[i + 1]);
      }
      if (textoOpcao == "--saidasubciclo" or textoOpcao == "-s") {
        saidaSubciclo = atoi(argv[i + 1]);
      }
    }
  }

  // Altera a GPU que será utilizada para execução. 
  hipSetDevice(idDevice);

  // Exclui a pasta de saída, se já existente. 
  system((EXCLUIR_PASTA + string("Saidas")).c_str());

  string entrada, saida;
  for (int idMC = 0; idMC < quantMCs; idMC++) {
    // O caminho para a pasta de entrada será "Entradas/MonteCarlo_{idMC}/"
    entrada = string("Entradas");
    entrada += SEP;
    entrada += string("MonteCarlo_");
    entrada += to_string(idMC);
    entrada += SEP;

    // O caminho para a pasta de saída será "Saidas/MonteCarlo_{idMC}/"
    saida = string("Saidas");
    saida += SEP;
    saida += string("MonteCarlo_");
    saida += to_string(idMC);
    saida += SEP;

    // Inicia a execução da simulação tipo Monte Carlo. 
    MonteCarlo(entrada, saida, saidaSubciclo);
  }

  return 0;
}
