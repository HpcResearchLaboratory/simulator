#include "Timer.h"

/*
  Classe responsável pelo cálculo do tempo de execução entre dois trechos de 
  código. Esta classe pode ser utilizada para mensurar o tempo gasto na 
  execução de métodos em GPU. O método "start" inicia a contagem do tempo, 
  "stop" termina a contagem do tempo e "getTime" retorna o tempo dispendido 
  em segundos. 
*/
Timer::Timer() {
  this->total = 0;
}

/*
  Método responsável por armazenar o tempo inicial da ocorrência de um evento. 
*/
void Timer::start() {
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord(begin);
}

/*
  Método responsável por armazenar o tempo final da ocorrência de um evento. 
  Com os tempos iniciais e finais é possível calcular o tempo dispendido em 
  uma operação. 
*/
void Timer::stop() {
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&parcial, begin, end);
  total += parcial;
}

/*
  Retorna o tempo calculado em segundos. 
*/
double Timer::getTime() {
  return total / 1000;
}
