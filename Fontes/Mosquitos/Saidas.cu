#include "hip/hip_runtime.h"
#include "Saidas.h"
#include "Fontes/Ambiente.h"
#include "Fontes/Saidas.h"
#include "Fontes/Mosquitos/Mosquitos.h"
#include "Fontes/Macros/MacrosMosquitos.h"
#include "Fontes/Macros/MacrosGerais.h"

/*
  Paralelizado para diferentes combinações de sexo, fase e saúde.
  
  Responsável pelo armazenamento e geração das saídas populacionais totais 
  para os agentes mosquitos, visão Dengue. A variável "popT" armazena os resultados 
  gerados pelo método "operator()". Esta classe é responsável pela geração dos 
  resultados armazenados no arquivo 
  "Saidas/MonteCarlo_{1}/Quantidades_Mosquitos_Dengue_Total.csv". 
*/
ContPopTMD::ContPopTMD(Mosquitos *mosquitos, Saidas *saidas, int ciclo) {
  this->mosquitos = mosquitos->PmosquitosDev;
  this->nMosquitos = mosquitos->nMosquitos;
  this->popT = saidas->PpopTMDDev;
  this->ciclo = ciclo;
}

/*
  Operador () da classe ContPopTMD.
*/
__host__ __device__
void ContPopTMD::operator()(int id) {
  for (int i = 0; i < nMosquitos; ++i) {
    // Agentes mortos não são contabilizados. 
    if (GET_VD_M(i) == MORTO) continue;
    
    // O id da coluna em que o agente será contabilizado depende de seu sexo, 
    // fase, saúde Dengue e sorotipo. 
    int desl = (GET_S_M(i) * N_FASES * N_ESTADOS_MD * (N_SOROTIPOS + 1));
    desl += (GET_FS_M(i) * N_ESTADOS_MD * (N_SOROTIPOS + 1));
    desl += ((GET_SD_M(i) - 1) * (N_SOROTIPOS + 1));
    desl += GET_ST_M(i);

    // Somente contabiliza o agente em uma coluna da saída. 
    if (desl == id) popT[VEC(ciclo, desl, N_COLS_MD)]++;
  }
}

/*
  Paralelizado para diferentes combinações de sexo, fase e saúde.
  
  Responsável pelo armazenamento e geração das saídas populacionais por 
  quadras para os agentes mosquitos, visão Dengue. A variável "popQ" armazena os 
  resultados gerados pelo método "operator()". A variável "indPopQ" armazena os 
  índices utilizados para indexar "popQ" por meio dos ids das quadras. Esta 
  classes é responsável pela geração dos resultados armazenados nos arquivos 
  "Saidas/MonteCarlo_{1}/Quantidades_Mosquitos_Dengue_Quadra-{2}.csv", em que 
  "{2}" é um id numérico para uma quadra. 
*/
ContPopQMD::ContPopQMD(Mosquitos *mosquitos, Saidas *saidas, int ciclo) {
  this->mosquitos = mosquitos->PmosquitosDev;
  this->nMosquitos = mosquitos->nMosquitos;
  this->indPopQ = saidas->PindPopQMDDev;
  this->popQ = saidas->PpopQMDDev;
  this->ciclo = ciclo;
}

/*
  Operador () da classe ContPopQMD.
*/
__host__ __device__
void ContPopQMD::operator()(int id) {
  for (int i = 0; i < nMosquitos; ++i) {
    // Agentes mortos não são contabilizados. 
    if (GET_VD_M(i) == MORTO) continue;
    
    // O id da coluna em que o agente será contabilizado depende de seu sexo, 
    // fase, saúde Dengue e sorotipo. 
    int desl = (GET_S_M(i) * N_FASES * N_ESTADOS_MD * (N_SOROTIPOS + 1));
    desl += (GET_FS_M(i) * N_ESTADOS_MD * (N_SOROTIPOS + 1));
    desl += ((GET_SD_M(i) - 1) * (N_SOROTIPOS + 1));
    desl += GET_ST_M(i);
    int q = GET_Q_M(i);

    // Somente contabiliza o agente em uma coluna da saída. 
    if (desl == id) popQ[indPopQ[q] + VEC(ciclo, desl, N_COLS_MD)]++;
  }
}

/*
  Paralelizado para diferentes combinações de sexo, fase e saúde.
  
  Responsável pelo armazenamento e geração das saídas espaciais para os 
  agentes mosquitos, visão Dengue. A variável "espacial" armazena os resultados 
  gerados pelo método "operator()". Esta classe é responsável pela geração dos 
  resultados armazenados no arquivo 
  "Saidas/MonteCarlo_{1}/Simulacao_{2}/Espacial_Mosquitos_Dengue.csv", em que 
  "{2}" é um id numérico para uma simulação individual. 
*/
ContEspacialMD::ContEspacialMD(
  Mosquitos *mosquitos, Saidas *saidas, Ambiente *ambiente,
  int nCiclos, int ciclo
) {
  this->mosquitos = mosquitos->PmosquitosDev;
  this->indMosquitos = mosquitos->PindMosquitosDev;
  this->espacial = saidas->PespacialMDDev;
  this->ciclo = ciclo;
  this->nCiclos = nCiclos;
  this->pos = ambiente->PposDev;
}

/*
  Operador () da classe ContEspacialMD.
*/
__host__ __device__
void ContEspacialMD::operator()(int id) {
  int x = pos[id].x, y = pos[id].y;
  int l = pos[id].lote, q = pos[id].quadra;
  int d = VEC(id, ciclo, nCiclos), e = 0, sd = 0, fs = 0;

  for (int i = indMosquitos[q]; i < indMosquitos[q + 1]; ++i) {
    // Somente agentes vivos nesta posição são representados na saída. 
    if (GET_VD_M(i) == MORTO or GET_L_M(i) != l or
        GET_X_M(i) != x or GET_Y_M(i) != y) continue;
  
    // Os ícones 4*** são utilizados para representar os agentes. 
    e = 4000;
    if (GET_FS_M(i) == ATIVA or GET_FS_M(i) == DECADENTE) {
      // Define a ordem de representação para os estados da Dengue. 
      switch (GET_SD_M(i)) {
        case INFECTANTE: sd = max(30, sd);
          break;
        case SUSCETIVEL: sd = max(20, sd);
          break;
        case EXPOSTO: sd = max(10, sd);
          break;
      }
    } else {
      // Se contém um ovo o final do identificador do ícone é 1. 
      fs = 1;
    }
  }
  espacial[d] = e + sd + fs;
}

/*
  Paralelizado para diferentes combinações de sexo, fase e saúde.
  
  Responsável pelo armazenamento e geração das saídas populacionais totais 
  para os agentes mosquitos, visão Wolbachia. A variável "popT" armazena os 
  resultados gerados pelo método "operator()". Esta classe é responsável pela 
  geração dos resultados armazenados no arquivo 
  "Saidas/MonteCarlo_{1}/Quantidades_Mosquitos_Wolbachia_Total.csv". 
*/
ContPopTMW::ContPopTMW(Mosquitos *mosquitos, Saidas *saidas, int ciclo) {
  this->mosquitos = mosquitos->PmosquitosDev;
  this->nMosquitos = mosquitos->nMosquitos;
  this->popT = saidas->PpopTMWDev;
  this->ciclo = ciclo;
}

/*
  Operador () da classe ContPopTMW.
*/
__host__ __device__
void ContPopTMW::operator()(int id) {
  for (int i = 0; i < nMosquitos; ++i) {
    // Agentes mortos não são contabilizados. 
    if (GET_VD_M(i) == MORTO) continue;
    
    // O id da coluna em que o agente será contabilizado depende de seu sexo, 
    // fase e saúde Wolbachia. 
    int desl = (GET_S_M(i) * N_FASES * N_ESTADOS_MW);
    desl += (GET_FS_M(i) * N_ESTADOS_MW);
    desl += GET_SW_M(i);

    // Somente contabiliza o agente em uma coluna da saída. 
    if (desl == id) popT[VEC(ciclo, desl, N_COLS_MW)]++;
  }
}

/*
  Paralelizado para diferentes combinações de sexo, fase e saúde.
  
  Responsável pelo armazenamento e geração das saídas populacionais por 
  quadras para os agentes mosquitos, visão Wolbachia. A variável "popQ" armazena 
  os resultados gerados pelo método "operator()". A variável "indPopQ" armazena 
  os índices utilizados para indexar "popQ" por meio dos ids das quadras. Esta 
  classe é responsável pela geração dos resultados armazenados nos arquivos 
  "Saidas/MonteCarlo_{1}/Quantidades_Mosquitos_Wolbachia_Quadra-{2}.csv", em que 
  "{2}" é um id numérico para uma quadra. 
*/
ContPopQMW::ContPopQMW(Mosquitos *mosquitos, Saidas *saidas, int ciclo) {
  this->mosquitos = mosquitos->PmosquitosDev;
  this->nMosquitos = mosquitos->nMosquitos;
  this->indPopQ = saidas->PindPopQMWDev;
  this->popQ = saidas->PpopQMWDev;
  this->ciclo = ciclo;
}

/*
  Operador () da classe ContPopQMW.
*/
__host__ __device__
void ContPopQMW::operator()(int id) {
  for (int i = 0; i < nMosquitos; ++i) {
    // Agentes mortos não são contabilizados. 
    if (GET_VD_M(i) == MORTO) continue;
    
    // O id da coluna em que o agente será contabilizado depende de seu sexo, 
    // fase e saúde Wolbachia. 
    int desl = (GET_S_M(i) * N_FASES * N_ESTADOS_MW);
    desl += (GET_FS_M(i) * N_ESTADOS_MW);
    desl += GET_SW_M(i);
    int q = GET_Q_M(i);

    // Somente contabiliza o agente em uma coluna da saída. 
    if (desl == id) popQ[indPopQ[q] + VEC(ciclo, desl, N_COLS_MW)]++;
  }
}

/*
  Paralelizado para diferentes combinações de sexo, fase e saúde.
  
  Responsável pelo armazenamento e geração das saídas populacionais não 
  acumuladas totais para os agentes mosquitos, visão Dengue. A variável "popNovoT" 
  armazena os resultados gerados pelo método "operator()". Esta classe é 
  responsável pela geração dos resultados armazenados no arquivo 
  "Saidas/MonteCarlo_{1}/Quantidades_Mosquitos_Dengue_Novo_Total.csv". 
*/
ContPopNovoTMD::ContPopNovoTMD(Mosquitos *mosquitos, Saidas *saidas, int ciclo) {
  this->mosquitos = mosquitos->PmosquitosDev;
  this->nMosquitos = mosquitos->nMosquitos;
  this->popNovoT = saidas->PpopNovoTMDDev;
  this->ciclo = ciclo;
}

/*
  Operador () da classe ContPopNovoTMD.
*/
__host__ __device__
void ContPopNovoTMD::operator()(int id) {
  for (int i = 0; i < nMosquitos; ++i) {
    // Agentes mortos não são contabilizados. 
    if (GET_VD_M(i) == MORTO) continue;
    // Agentes machos não são contabilizados.
    if (GET_S_M(i) == MACHO) continue;
    // Fêmeas não acasaladas não são contabilizadas.
    if (GET_TA_M(i) == NENHUM) continue;
    // Somente agentes infectados são contabilizados.
    if (GET_SD_M(i) != INFECTANTE) continue;
    // Somente agentes com o contador = 0 são contabilizados. 
    if (GET_C_M(i) != 0) continue;

    // O id da coluna em que o agente será contabilizado depende de seu sexo, 
    // fase, saúde Dengue e sorotipo. 
    int desl = (GET_S_M(i) * N_FASES * N_ESTADOS_MD * (N_SOROTIPOS + 1));
    desl += (GET_FS_M(i) * N_ESTADOS_MD * (N_SOROTIPOS + 1));
    desl += ((GET_SD_M(i) - 1) * (N_SOROTIPOS + 1));
    desl += GET_ST_M(i);

    // Somente contabiliza o agente em uma coluna da saída. 
    if (desl == id) popNovoT[VEC(ciclo, desl, N_COLS_MD)]++;
  }
}

/*
  Paralelizado para diferentes combinações de sexo, fase e saúde.
  
  Responsável pelo armazenamento e geração das saídas populacionais não 
  acumuladas por quadras para os agentes mosquitos, visão Dengue. A variável 
  "popQ" armazena os resultados gerados pelo método "operator()". Esta classe 
  é responsável pela geração dos resultados armazenados nos arquivos 
  "Saidas/MonteCarlo_{1}/Quantidades_Mosquitos_Dengue_Novo_Quadra-{2}.csv", em 
  que "{2}" é um id numérico para uma quadra. 
*/
ContPopNovoQMD::ContPopNovoQMD(Mosquitos *mosquitos, Saidas *saidas, int ciclo) {
  this->mosquitos = mosquitos->PmosquitosDev;
  this->nMosquitos = mosquitos->nMosquitos;
  this->indPopQ = saidas->PindPopQMDDev;
  this->popQ = saidas->PpopNovoQMDDev;
  this->ciclo = ciclo;
}

/*
  Operador () da classe ContPopNovoQMD.
*/
__host__ __device__
void ContPopNovoQMD::operator()(int id) {
  for (int i = 0; i < nMosquitos; ++i) {
    // Agentes mortos não são contabilizados. 
    if (GET_VD_M(i) == MORTO) continue;
    // Somente agentes que mudaram de estado há um ciclo são contabilizados. 
    if (GET_C_M(i) != 1) continue;
    
    // O id da coluna em que o agente será contabilizado depende de seu sexo, 
    // fase, saúde Dengue e sorotipo. 
    int desl = (GET_S_M(i) * N_FASES * N_ESTADOS_MD * (N_SOROTIPOS + 1));
    desl += (GET_FS_M(i) * N_ESTADOS_MD * (N_SOROTIPOS + 1));
    desl += ((GET_SD_M(i) - 1) * (N_SOROTIPOS + 1));
    desl += GET_ST_M(i);
    int q = GET_Q_M(i);

    // Somente contabiliza o agente em uma coluna da saída. 
    if (desl == id) popQ[indPopQ[q] + VEC(ciclo, desl, N_COLS_MD)]++;
  }
}