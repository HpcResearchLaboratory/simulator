#include "hip/hip_runtime.h"
#include "Simulacao.h"

#include "Fontes/Seeds.h"
#include "Fontes/Parametros.h"
#include "Fontes/Ambiente.h"
#include "Fontes/Uteis/RandPerc.h"
#include "Fontes/Saidas.h"
#include "Fontes/Macros/MacrosSO.h"
#include "Fontes/Macros/MacrosGerais.h"
#include "Fontes/Macros/2_CON_H.h"
#include "Fontes/Macros/3_TRA_H.h"
#include "Fontes/Macros/4_CON_H.h"
#include "Fontes/Macros/3_TRA_M.h"
#include "Fontes/Macros/4_CON_M.h"
#include "Fontes/Macros/5_GER_M.h"

#include "Fontes/Mosquitos/Mosquitos.h"
#include "Fontes/Mosquitos/Movimentacao.h"
#include "Fontes/Mosquitos/Contato.h"
#include "Fontes/Mosquitos/Transicao.h"
#include "Fontes/Mosquitos/Controle.h"
#include "Fontes/Mosquitos/Geracao.h"
#include "Fontes/Mosquitos/Insercao.h"
#include "Fontes/Mosquitos/Saidas.h"

#include "Fontes/Humanos/Humanos.h"
#include "Fontes/Humanos/Movimentacao.h"
#include "Fontes/Humanos/Contato.h"
#include "Fontes/Humanos/Transicao.h"
#include "Fontes/Humanos/Controle.h"
#include "Fontes/Humanos/Insercao.h"
#include "Fontes/Humanos/Saidas.h"

#include <chrono>

using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::microseconds;

/*
  Construtor da classe Simulacao.

  A variável "idSim" indica o id numérico da simulação individual.
  "saidaSim" indica o caminho para a pasta de saída da simulação.
  "saidas", "parametros" e "ambiente" armazenam as saídas, os parâmetros e o
  ambiente de simulação, respectivamente.

  Este método é responsável por criar a pasta de saída dos arquivos resultantes
  da simulação, inicializar as populações de humanos e mosquitos, inicializar
  as seeds utilizadas à geração de números aleatórios, exibir em tela o
  consumo de memória total da simulação, iniciar a execução da simulação
  individual, copiar os resultados da simulação da GPU para a mémoria
  principal e salvar as saídas espaciais da simulação. Note que somente as
  saídas espaciais são salvas para a simulação individual. As saídas
  populacionais são tipo Monte Carlo e são salvas pela classe MonteCarlo.
*/
Simulacao::Simulacao(
  int idSim, string saidaSim, Saidas *saidas,
  Parametros *parametros, Ambiente *ambiente,
  int saidaSubciclo
) {
  this->idSim = idSim;
  this->saidaSim = saidaSim;
  this->saidas = saidas;
  this->parametros = parametros;
  this->ambiente = ambiente;
  this->saidaSubciclo = saidaSubciclo;

  ciclo = 0;
  periodo = MANHA;
  subciclo = 0;
  idLira = -1;

  // As pastas de saída bistring serão:
  // "Entradas/MonteCarlo_{idMC}/Simulacao_{idSim}/Mosquitos/".
  // "Entradas/MonteCarlo_{idMC}/Simulacao_{idSim}/Humanos/".
  saidaM = saidaSim + string("Mosquitos") + SEP;
  saidaH = saidaSim + string("Humanos") + SEP;

  // O arquivo de saída para oviposição será:
  // "Entradas/MonteCarlo_{idMC}/Simulacao_{idSim}/Oviposicao.csv".
  arquivoSaidaOviposicao = saidaSim + string("Oviposicao.csv");

  // Criação da pasta de saída da simulação individual.
  system((CRIAR_PASTA + saidaSim).c_str());

  // Criação das pastas de saída bitstring.
  system((CRIAR_PASTA + saidaM).c_str());
  system((CRIAR_PASTA + saidaH).c_str());

  // Criação dos agentes humanos e mosquitos.
  humanos = new Humanos(parametros, ambiente);
  mosquitos = new Mosquitos(parametros, ambiente);

  // Inicialização das seeds.
  seeds = new Seeds(
                {mosquitos->maxMosquitos,
                humanos->nHumanos, ambiente->sizePos}
              );

  // Exibição em tela do consumo de memória total da simulação individual.
  if (idSim == 0) exibirConsumoMemoria();

  // Inicialização da execução da simulação índividual.
  iniciar();

  // Cópia das saídas da simulação que estão em GPU para a CPU.
  saidas->toCPU();
  // Escrita dos arquivos de saída espaciais da simulação individual.
  saidas->salvarEspaciais(saidaSim);
}

/*
  Destrutor da classe Simulacao.

  São desalocados as classes que armazenam os agentes humanos e mosquitos e as
  seeds utilizadas durante a simulação.
*/
Simulacao::~Simulacao() {
  delete(humanos); delete(mosquitos); delete(seeds);
}

/*
  Método responsável por executar o processo de simulação. São executados os
  operadores definidos à modelagem da Dengue na ordem especificada. O primeiro
  for é responsável por executar os ciclos de simulação. O segundo for é
  responsável pela execução dos períodos dos ciclos. O terceiro for é
  responsável por executar os subciclos correspondentes aos períodos.
  A movimentação dos humanos é realizada uma vez a cada período. A movimentação
  dos mosquitos e os contatos entre mosquitos e entre mosquitos e humanos são
  realizadas uma vez a cada subciclo. Os demais operadores são executados
  uma vez a cada ciclo.
*/
void Simulacao::iniciar() {

  // Executa movimentação, contato e transição para estabilizar a população.
  int CICLOS_SHIFT_POPULACAO = 0;
  for (ciclo = 1; ciclo < (CICLOS_SHIFT_POPULACAO + 1); ++ciclo) {
    // cout << "Inicializacao | ";
    // cout << "Ciclo " << ciclo << " / " << CICLOS_SHIFT_POPULACAO << endl;
    for (periodo = MANHA; periodo <= NOITE; ++periodo) {
      cout << "periodo: " << periodo << endl;
      movimentacaoHumanos();
      for (subciclo = 0; subciclo < parametros->nSubCiclos; ++subciclo) {
        // cout << "subciclo: " << subciclo << endl;
        movimentacaoMosquitos();
        contatoEntreMosquitos(periodo);
        contatoEntreMosquitosEHumanos(periodo);
      }
    }
    transicaoFasesMosquitos();
    transicaoEstadosMosquitos();
    transicaoEstadosHumanos();
    controleNaturalMosquitosPorIdade();
    controleNaturalMosquitosPorSelecao();
    controleNaturalHumanos();
    geracao();
  }

  ciclo = 0;

  // Execução dos ciclos de simulação.
  for (ciclo = 1; ciclo < parametros->nCiclos; ++ciclo) {
    // cout << "Ciclo " << ciclo << " / " << (parametros->nCiclos - 1) << endl;
    // calcularRt();
    calcularIdLira();
    computarSaidas();
    saidaOviposicao();
    
    // Execução dos períodos do ciclo.
    for (periodo = MANHA; periodo <= NOITE; ++periodo) {
      if (saidaSubciclo or periodo == MANHA)
        saidaBitstringHumanos();
      
      // cout << "periodo: " << periodo << endl;
      movimentacaoHumanos(); // Ok
      
      // Execução dos subciclos do período.
      for (subciclo = 0; subciclo < parametros->nSubCiclos; ++subciclo) {
        if (saidaSubciclo or (periodo == MANHA and subciclo == 0))
          saidaBitstringMosquitos();
        
        // cout << "subciclo: " << subciclo << endl;
        movimentacaoMosquitos(); // Ok

        contatoEntreMosquitos(periodo); // Ok
        contatoEntreMosquitosEHumanos(periodo); // Ok
      }
    }

    transicaoFasesMosquitos();
    transicaoEstadosMosquitos(); // Ok
    transicaoEstadosHumanos(); // Ok
    //vacinacao();

    controleNaturalMosquitosPorIdade(); // Ok
    controleNaturalMosquitosPorSelecao(); // Ok
    controleNaturalHumanos();
    //controlesMosquitos();
    //controleBiologico();
    //tratamentoAmbiental();

    geracao(); // Ok

    insercaoMosquitos(); // Ok
    insercaoHumanos(); // Ok

  }
  periodo = 0;
  subciclo = 0;
  saidaBitstringHumanos();
  saidaBitstringMosquitos();
  computarSaidas();
  saidaOviposicao();
}

/*
void Simulacao::calcularRt() {
  RandPerc rand;

  // Parâmetros para humanos
  double beta_h = TAXA_INFECCAO_MOSQUITO_(rand());
  int S = count_if(
    humanos->humanosDev->begin(),
    humanos->humanosDev->end(),
    HumanoSuscetivel()
  );
  int N = humanos->nHumanos - count_if(
    humanos->humanosDev->begin(),
    humanos->humanosDev->end(),
    EstaMortoHumano()
  );
  double gamma_h = (PERIODO_EXPOSTO_HUMANO_(CRIANCA, rand()) +
                    PERIODO_EXPOSTO_HUMANO_(JOVEM, rand()) +
                    PERIODO_EXPOSTO_HUMANO_(ADULTO, rand()) +
                    PERIODO_EXPOSTO_HUMANO_(IDOSO, rand())) / 4.0;
  double mi_h = TAXA_MORTE_NATURAL_(rand());
  double sigma_h = 1.0 / ((PERIODO_RECUPERADO_HUMANO_(CRIANCA, rand()) +
                           PERIODO_RECUPERADO_HUMANO_(JOVEM, rand()) +
                           PERIODO_RECUPERADO_HUMANO_(ADULTO, rand()) +
                           PERIODO_RECUPERADO_HUMANO_(IDOSO, rand())) / 4.0);

  // Parâmetros para mosquitos
  double beta_m = (TAXA_INFECCAO_HUMANO_SUSCETIVEL_(CRIANCA, rand()) +
                   TAXA_INFECCAO_HUMANO_SUSCETIVEL_(JOVEM, rand()) +
                   TAXA_INFECCAO_HUMANO_SUSCETIVEL_(ADULTO, rand()) +
                   TAXA_INFECCAO_HUMANO_SUSCETIVEL_(IDOSO, rand())) / 4.0;
  int m1 = count_if(
    mosquitos->mosquitosDev->begin(),
    mosquitos->mosquitosDev->end(),
    MosquitoFemeaSuscetivelAlado()
  );
  int m = count_if(
    mosquitos->mosquitosDev->begin(),
    mosquitos->mosquitosDev->end(),
    MosquitoFemeaAlado()
  );
  double gamma_m = 1.0 / (CICLOS_LATENCIA_MOSQUITOS_(rand()));
  double mi_f = BS_ATIVOS_(rand()) *
                TAXA_ELIM_CONTROLE_NATURAL_SELECAO_(ATIVA, FEMEA, SAUDAVEL, rand()) *
                ENTRE_FAIXA(ambiente->cli[ciclo].txMinAlados, ambiente->cli[ciclo].txMaxAlados, rand());
  double fi = AS21_(rand()) / (double) (INTERVALO_ENTRE_POSTURAS_FEMEA_(rand()) +
                                        CICLOS_GESTACAO_(rand()));

  // I(t) e I(t + 1)
  double casos_t = ambiente->casos[ciclo - 1];
  double casos_t_add1 = ambiente->casos[ciclo];

  // Cálculo do ref
  if (casos_t != 0) {
    ref = casos_t_add1 / casos_t;
  }

  // Variáveis para o cálculo de q0 e m_star
  double sigma_p = BS_PUPAS_(rand());
  double sigma_l = BS_LARVAS_(rand());
  double mi_p = 1.0 - sigma_p;
  double mi_l = 1.0 - sigma_l;
  //double C = (ambiente->indPosReg[3] - ambiente->indPosReg[2]) * ENTRE_FAIXA(20, 30, rand());
  double C = 74 * ENTRE_FAIXA(20, 30, rand());
  double qf = (1.0 - PS21_(rand())) * BS_OVOS_(rand());

  // Cálculo do Q0
  double a = (sigma_l / (sigma_l + mi_l));
  double b = (sigma_p / (sigma_p + mi_p));
  double c = (qf * fi / mi_f);
  double q0 = (sigma_l * sigma_p * qf * fi) / ((sigma_l + mi_l) * (sigma_p + mi_p) * mi_f);
  double m_star = (sigma_p * sigma_l * C * (1 - 1 / q0)) / (mi_f * (sigma_p + mi_p));

  beta_h = (ref * m * pow(N, 2.0) * (gamma_m + mi_f) * (gamma_h + mi_h) * (sigma_h + mi_h) * mi_f) /
           (pow(fi, 2.0) * gamma_h * gamma_m * S * m1 * m_star);
  beta_h /= beta_m;

  std::cout << ref << "\t" << q0 << "\t" << beta_h << std::endl;
}
*/

/*
  Método responsável por calcular o LIRAa atual de acordo com o ciclo,
  determinando assim os multiplicadores aplicados no controle de ovos.
*/
void Simulacao::calcularIdLira() {
  double frac = (double) ciclo / parametros->nCiclos;
  idLira = (int) (frac * ambiente->nLira);
}

/*
  Método responsável pela movimentação dos agentes humanos.

  O método "for_each_n" é responsável pela aplicação do operador
  "MovimentacaoHumanos" sobre toda a população de agentes humanos. Como a
  biblioteca Thrust é utilizada, a aplicação desta operação pode ocorrer
  paralelamente sobre os dados, dependendo das flags utilizadas durante a
  compilação realizada.

  O método "humanos->atualizacaoIndices" é responsável pela atualização dos
  índices da estrutura que armazena os agentes humanos. Este índice agiliza
  a obtenção dos humanos que estão em uma determinada localidade. Por exemplo,
  "indHumanos[10]" armazena a primeira posição da região de dados que contém os
  agentes posicionados na localidade "10". A atualização dos índices é necessária
  pois a movimentação pode alterar a localidade em que os humanos estão posicionados.
*/
void Simulacao::movimentacaoHumanos() {
  // auto start = high_resolution_clock::now();
  for_each_n(
    seeds->idx, humanos->nHumanos,
    MovimentacaoHumanos(humanos, ambiente, parametros, seeds)
  );
  humanos->atualizacaoIndices();
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "movimentacaoHumanos(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pela movimentação dos agentes mosquitos.

  O método "for_each_n" é responsável pela aplicação do operador
  "MovimentacaoMosquitos" sobre toda a população de agentes mosquitos. Como a
  biblioteca Thrust é utilizada, a aplicação desta operação pode ocorrer
  paralelamente sobre os dados, dependendo das flags utilizadas durante a
  compilação realizada.

  O método "mosquitos->atualizacaoIndices" é responsável pela atualização dos
  índices da estrutura que armazena os agentes mosquitos. Este índice agiliza
  a obtenção dos mosquitos que estão em uma determinada localidade. Por exemplo,
  "indMosquitos[10]" armazena a primeira posição da região de dados que contém
  os agentes posicionados na localidade "10". A atualização dos índices é necessária
  pois a movimentação pode alterar a localidade em que os mosquitos estão
  posicionados.
*/
void Simulacao::movimentacaoMosquitos() {
  // auto start = high_resolution_clock::now();
  for_each_n(
    seeds->idx, mosquitos->nMosquitos,
    MovimentacaoMosquitos(
      mosquitos, humanos, ambiente,
      parametros, periodo, ambiente->sizePontEst, seeds
    )
  );
  mosquitos->atualizacaoIndices();
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "movimentacaoMosquitos(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pelo contato entre agentes mosquitos, em que ocorrem
  os acasalamentos.

  O método "for_each_n" é responsável pela aplicação do operador
  "ContatoMosquitos" sobre todo o ambiente de simulação. Como a biblioteca
  Thrust é utilizada, a aplicação desta operação pode ocorrer paralelamente
  sobre os dados, dependendo das flags utilizadas durante a compilação realizada.
*/
void Simulacao::contatoEntreMosquitos(int periodo) {
  // auto start = high_resolution_clock::now();
  for_each_n(
    seeds->idx, ambiente->sizePos,
    ContatoMosquitos(mosquitos, ambiente, parametros, periodo, seeds)
  );
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "contatoEntreMosquitos(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pelo contato entre agentes mosquitos e humanos, em que
  ocorrem a transmissão da doença de agentes infectados para agentes suscetíveis.

  O método "for_each_n" é responsável pela aplicação do operador
  "ContatoHumanos" sobre todo o ambiente de simulação. Como a biblioteca
  Thrust é utilizada, a aplicação desta operação pode ocorrer paralelamente
  sobre os dados, dependendo das flags utilizadas durante a compilação realizada.
*/
void Simulacao::contatoEntreMosquitosEHumanos(int periodo) {
  // auto start = high_resolution_clock::now();
  for_each_n(
    seeds->idx, ambiente->sizePos,
    ContatoHumanos(mosquitos, humanos, ambiente, parametros, ciclo,
                   periodo, seeds)
  );
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "contatoEntreMosquitosEHumanos(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pela transição de fases dos agentes mosquitos.

  O método "for_each_n" é responsável pela aplicação do operador
  "TransicaoFasesMosquitos" sobre todo o ambiente de simulação.
  Como a biblioteca Thrust é utilizada, a aplicação desta operação pode ocorrer
  paralelamente sobre os dados, dependendo das flags utilizadas durante a
  compilação realizada.
*/
void Simulacao::transicaoFasesMosquitos() {
  // auto start = high_resolution_clock::now();
  for_each_n(
    seeds->idx, ambiente->sizePos,
    TransicaoFasesMosquitos(ambiente, mosquitos, parametros, seeds)
  );
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "transicaoFasesMosquitos(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pela transição de estados dos agentes mosquitos, em que
  ocorre a evolução da doença dos agentes infectados.

  O método "for_each_n" é responsável pela aplicação do operador
  "TransicaoEstadosMosquitos" sobre toda a população de agentes mosquitos.
  Como a biblioteca Thrust é utilizada, a aplicação desta operação pode ocorrer
  paralelamente sobre os dados, dependendo das flags utilizadas durante a
  compilação realizada.
*/
void Simulacao::transicaoEstadosMosquitos() {
  // auto start = high_resolution_clock::now();
  for_each_n(
    seeds->idx, mosquitos->nMosquitos,
    TransicaoEstadosMosquitos(mosquitos, parametros, seeds)
  );
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "transicaoEstadosMosquitos(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pela transição de estados dos agentes humanos, em que
  ocorre a evolução da doença dos agentes infectados.

  O método "for_each_n" é responsável pela aplicação do operador
  "TransicaoEstadosHumanos" sobre toda a população de agentes humanos. Como a
  biblioteca Thrust é utilizada, a aplicação desta operação pode ocorrer
  paralelamente sobre os dados, dependendo das flags utilizadas durante a
  compilação realizada.
*/
void Simulacao::transicaoEstadosHumanos() {
  // auto start = high_resolution_clock::now();
  for_each_n(
    seeds->idx, humanos->nHumanos,
    TransicaoEstadosHumanos(humanos, parametros, seeds)
  );
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "transicaoEstadosHumanos(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pela vacinação dos agentes humanos.

  A primeira chamada ao método "for_each_n" é responsável pela aplicação do
  operador "CampanhaVacinacao" sobre todas as localidades que em serão aplicadas
  campanhas de vacinação.

  A segunda chamada ao método "for_each_n" é responsável pela aplicação do
  operador "PosCampanhaVacinacao", que realiza a atualização da campanha de
  vacinação ao longo do tempo.

  A terceira chamado ao método "for_each_n" é responsável pela aplicação do
  operador "InsercaoVacinados", que realiza a inserção de humanos vacinados
  no ambiente. Efetivamente novos humanos não são inseridos, ocorre somente a
  troca de estados de humanos já existentes na simulação.

  Como a biblioteca Thrust é utilizada, a aplicação destas operações podem
  ocorrer paralelamente sobre os dados, dependendo das flags utilizadas durante
  a compilação realizada.
*/
void Simulacao::vacinacao() {
  for_each_n(
    seeds->idx, ambiente->sizeQuadVac,
    CampanhaVacinacao(
      humanos, ambiente, parametros, ciclo,
      ambiente->sizeQuadVac, ambiente->sizeFEVac,
      ambiente->sizePerVac, ambiente->sizeCicVac, seeds
    )
  );
  for_each_n(
    seeds->idx, 1,
    PosCampanhaVacinacao(
      ambiente, ciclo, ambiente->sizePerVac, ambiente->sizeCicVac
    )
  );

  for_each_n(
    seeds->idx, ambiente->sizeVacs,
    InsercaoVacinados(
      humanos, ambiente, parametros, ciclo, seeds
    )
  );
}

/*
  Método responsável pela execução do controle natural por idade sobre a
  população de mosquitos.

  O método "for_each_n" é responsável pela aplicação do operador
  "ControleNaturalMosquitosPorIdade" sobre toda a população de agentes mosquitos.
  Como a biblioteca Thrust é utilizada, a aplicação desta operação pode ocorrer
  paralelamente sobre os dados, dependendo das flags utilizadas durante a
  compilação realizada.
*/
void Simulacao::controleNaturalMosquitosPorIdade() {
  // auto start = high_resolution_clock::now();
  for_each_n(
    seeds->idx, mosquitos->nMosquitos,
    ControleNaturalMosquitosPorIdade(mosquitos, parametros, seeds)
  );
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "controleNaturalMosquitosPorIdade(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pela execução do controle natural por seleção sobre a
  população de mosquitos.

  O método "for_each_n" é responsável pela aplicação do operador
  "ControleNaturalMosquitosPorSelecao" sobre todo o ambiente de simulação. Como
  a biblioteca Thrust é utilizada, a aplicação desta operação pode ocorrer
  paralelamente sobre os dados, dependendo das flags utilizadas durante a
  compilação realizada.
*/
void Simulacao::controleNaturalMosquitosPorSelecao() {
  // auto start = high_resolution_clock::now();
  for_each_n(
    seeds->idx, ambiente->sizePos,
    ControleNaturalMosquitosPorSelecao(
      mosquitos, ambiente, parametros, ciclo, idLira, seeds
    )
  );
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "controleNaturalMosquitosPorSelecao(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pela execução do controle natural sobre a população de
  humanos.

  O método "for_each_n" é responsável pela aplicação do operador
  "ControleNaturalHumanos" sobre toda a população de agentes humanos. Como
  a biblioteca Thrust é utilizada, a aplicação desta operação pode ocorrer
  paralelamente sobre os dados, dependendo das flags utilizadas durante a
  compilação realizada.
*/
void Simulacao::controleNaturalHumanos() {
  // auto start = high_resolution_clock::now();
  for_each_n(
    seeds->idx, humanos->nHumanos,
    ControleNaturalHumanos(humanos, parametros, seeds)
  );
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "controleNaturalHumanos(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pela execução dos controles sobre a população de mosquitos
  no ambiente de simulação.

  O operador "PreControlesMosquitos" é responsável por verificar se ocorrerá a
  aplicação dos controles neste ciclo de simulação. Em caso afirmativo,
  os percentuais dos efeitos residuais são alterados para o valor máximo,
  indicando que os controles químicos têm eficiência inicial máxima de acordo
  com a taxa definida nos arquivos de configuração dos controles.

  A segunda chamada ao método "for_each_n" é responsável pela aplicação do
  operador "ControlesQuimicos" sobre todo o ambiente de simulação. Este operador
  executa os controles tipo Raio, Bloqueio, Tratamento e Pontos Estratégicos.
  Como a biblioteca Thrust é utilizada, a aplicação desta operação pode ocorrer
  paralelamente sobre os dados, dependendo das flags utilizadas durante a
  compilação realizada.

  Por fim, o operador "PosControlesMosquitos" é responsável por decrementar os
  percentuais dos efeitos residuais de acordo com os parâmetros especificados
  para o particular controle, indicando que os efeitos dos controles químicos
  aplicados decrescem ao longo do tempo.
*/
void Simulacao::controlesMosquitos() {
  for (int idContr = 0; idContr < ambiente->sizeContr and
       ambiente->contr[idContr].ciclo <= ciclo; ++idContr) {
    for_each_n(
      seeds->idx, 1,
      PreControlesMosquitos(
        ambiente, parametros, ciclo, idContr, seeds
      )
    );

    for_each_n(
      seeds->idx, ambiente->sizePos,
      ControlesMosquitos(
        mosquitos, ambiente, parametros, ciclo,
        idContr, ambiente->sizePontEst, seeds
      )
    );

    for_each_n(
      seeds->idx, 1,
      PosControlesMosquitos(
        ambiente, parametros, idContr, seeds
      )
    );
  }
}

/*
  Método responsável pela execução do controle biológico durante a simulação,
  que realiza a inserção de agentes mosquitos infectados com Wolbachia no
  ambiente.

  Inicialmente é obtida a quantidade total de agentes mosquitos que serão
  inseridos no ambiente. Esta quantidade depende dos parâmetros definidos nos
  arquivos de configuração.

  Em seguida são inseridos os agentes mosquitos. Os novos agentes são inseridos,
  se possível, em posições do vetor de agentes mosquitos que contenham agentes
  mortos, com o objetivo de otimizar o uso de memória e evitar realocações
  desnecessárias. O vetor de mosquitos somente é realocado se a quantidade de
  agentes que serão inseridos é maior que a quantidade de agentes mortos.
  Antes da inserção o vetor de agentes é particionado, movendo os agentes
  mortos para o início do vetor, facilitando desta forma a inserção dos novos
  agentes. For fim são atualizados os índices para os mosquitos, pois as
  quantidades de agentes nas localidades foram alterados.

  O método "for_each_n" é responsável pela aplicação do operador
  "ControleBiologico" sobre todas as localidades onde serão inseridos agentes
  mosquitos infectados com Wolbachia. Como a biblioteca Thrust é utilizada,
  a aplicação desta operação pode ocorrer paralelamente sobre os dados,
  dependendo das flags utilizadas durante a compilação realizada.
*/
void Simulacao::controleBiologico() {
  int n = transform_reduce(
            seeds->idx, seeds->idx + ambiente->sizeConBio,
            PreControleBiologico(parametros, ciclo),
            0, plus<int>()
          );
  if (n > 0) {
    int m = count_if(
              mosquitos->mosquitosDev->begin(),
              mosquitos->mosquitosDev->end(),
              EstaMortoMosquito()
            );

    if (n > m) {
      if (mosquitos->alocarMosquitos and
          (mosquitos->nMosquitos + (n - m)) > mosquitos->maxMosquitos) {
        cout << "Quantidade maxima de mosquitos alcancada ";
        cout << "no ciclo " << ciclo << endl;
        mosquitos->alocarMosquitos = false;
      }
      if (mosquitos->alocarMosquitos) {
        mosquitos->nMosquitos += (n - m);
        mosquitos->mosquitosDev->resize(mosquitos->nMosquitos, Mosquito());
        mosquitos->PmosquitosDev =
          raw_pointer_cast(mosquitos->mosquitosDev->data());
      }
    }

    partition(
      mosquitos->mosquitosDev->begin(),
      mosquitos->mosquitosDev->end(),
      EstaMortoMosquito()
    );

    for_each_n(
      seeds->idx, ambiente->sizeConBio,
      ControleBiologico(
        mosquitos, ambiente, parametros, ciclo, mosquitos->alocarMosquitos,
        seeds
      )
    );

    mosquitos->atualizacaoIndices();
  }
}

/*
  Método responsável pela execução do tratamento ambiental sobre o ambiente de
  simulação.

  O método "for_each_n" é responsável pela aplicação do operador
  "TratamentoAmbiental" sobre todas as quadras em que serão executadas o
  tratamento ambiental. Como a biblioteca Thrust é utilizada, a aplicação desta
  operação pode ocorrer paralelamente sobre os dados, dependendo das flags
  utilizadas durante a compilação realizada.
*/
void Simulacao::tratamentoAmbiental() {
  for_each_n(
    seeds->idx, ambiente->sizeConAmb,
    TratamentoAmbiental(
      mosquitos, ambiente, parametros, ciclo, ambiente->sizeConAmb, seeds
    )
  );
}

/*
  Método responsável pela execução da geração durante a simulação, que realiza
  a inserção de agentes mosquitos ovos resultantes dos contatos entre agentes
  mosquitos alados.

  Inicialmente é obtida a quantidade total de agentes mosquitos ovos que serão
  inseridos. Esta quantidade depende dos parâmetros definidos nos arquivos de
  configuração e dos contatos realizados entre agentes mosquitos neste ciclo.

  Em seguida são inseridos os agentes mosquitos. Os novos agentes são inseridos,
  se possível, em posições do vetor de agentes mosquitos que contenham agentes
  mortos, com o objetivo de otimizar o uso de memória e evitar realocações
  desnecessárias. O vetor de mosquitos somente é realocado se a quantidade de
  agentes que serão inseridos é maior que a quantidade de agentes mortos.
  Antes da inserção o vetor de agentes é particionado, movendo os agentes
  mortos para o início do vetor, facilitando desta forma a inserção dos novos
  agentes. For fim são atualizados os índices para os mosquitos, pois as
  quantidades de agentes nas localidades foram alterados.

  O método "for_each_n" é responsável pela aplicação do operador
  "Geracao" para a inserção dos novos agentes mosquitos ovos.
*/
void Simulacao::geracao() {
  // auto start = high_resolution_clock::now();
  int n = transform_reduce(
            seeds->idx, seeds->idx + mosquitos->nMosquitos,
            PreGeracao(ambiente, mosquitos, parametros, seeds),
            0, plus<int>()
          );
  if (n > 0) {
    int m = count_if(
              mosquitos->mosquitosDev->begin(),
              mosquitos->mosquitosDev->end(),
              EstaMortoMosquito()
            );

    if (n > m) {
      if (mosquitos->alocarMosquitos and
          (mosquitos->nMosquitos + (n - m)) > mosquitos->maxMosquitos) {
        cout << "Quantidade maxima de mosquitos alcancada ";
        cout << "no ciclo " << ciclo << endl;
        mosquitos->alocarMosquitos = false;
      }
      if (mosquitos->alocarMosquitos) {
        mosquitos->nMosquitos += (n - m);
        mosquitos->mosquitosDev->resize(mosquitos->nMosquitos, Mosquito());
        mosquitos->PmosquitosDev =
          raw_pointer_cast(mosquitos->mosquitosDev->data());
      }
    }

    partition(
      mosquitos->mosquitosDev->begin(),
      mosquitos->mosquitosDev->end(),
      EstaMortoMosquito()
    );

    for_each_n(
      seeds->idx, 1,
      Geracao(
        ambiente, mosquitos, parametros, seeds
      )
    );

    mosquitos->atualizacaoIndices();
  }
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "geracao(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pela execução da inserção de agentes mosquitos no ambiente
  durante a simulação.

  Inicialmente é obtida a quantidade total de agentes mosquitos que serão
  inseridos. Esta quantidade depende dos parâmetros definidos nos arquivos de
  configuração, principalmente do arquivo "DistribuicaoMosquitos.csv".

  Em seguida são inseridos os agentes mosquitos. Os novos agentes são inseridos,
  se possível, em posições do vetor de agentes mosquitos que contenham agentes
  mortos, com o objetivo de otimizar o uso de memória e evitar realocações
  desnecessárias. O vetor de mosquitos somente é realocado se a quantidade de
  agentes que serão inseridos é maior que a quantidade de agentes mortos.
  Antes da inserção o vetor de agentes é particionado, movendo os agentes
  mortos para o início do vetor, facilitando desta forma a inserção dos novos
  agentes. For fim são atualizados os índices para os mosquitos, pois as
  quantidades de agentes nas localidades foram alterados.

  O método "for_each_n" é responsável pela aplicação do operador
  "InsercaoMosquitos" para a inserção dos novos agentes mosquitos.
*/
void Simulacao::insercaoMosquitos() {
  // auto start = high_resolution_clock::now();
  int n = transform_reduce(
            seeds->idx, seeds->idx + 1,
            PreInsercaoMosquitos(ciclo, ambiente), 0, plus<int>()
          );
  if (n > 0) {
    int m = count_if(
              mosquitos->mosquitosDev->begin(),
              mosquitos->mosquitosDev->end(),
              EstaMortoMosquito()
            );

    if (n > m) {
      mosquitos->nMosquitos += (n - m);
      mosquitos->mosquitosDev->resize(mosquitos->nMosquitos, Mosquito());
      mosquitos->PmosquitosDev =
                            raw_pointer_cast(mosquitos->mosquitosDev->data());
    }

    partition(
      mosquitos->mosquitosDev->begin(),
      mosquitos->mosquitosDev->end(),
      EstaMortoMosquito()
    );

    for_each_n(
      seeds->idx, 1,
      InsercaoMosquitos(mosquitos, ambiente, parametros, ciclo,
                        ambiente->sizePontEst, seeds)
    );

    mosquitos->atualizacaoIndices();
  }
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "insercaoMosquitos(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pela execução da inserção de agentes humanos no ambiente
  durante a simulação.

  Inicialmente é obtida a quantidade total de agentes humanos que serão
  inseridos. Esta quantidade depende dos parâmetros definidos nos arquivos de
  configuração, principalmente do arquivo "DistribuicaoHumanos.csv".

  Em seguida são inseridos os agentes humanos. Os novos agentes são inseridos,
  se possível, em posições do vetor de agentes humanos que contenham agentes
  mortos, com o objetivo de otimizar o uso de memória e evitar realocações
  desnecessárias. O vetor de humanos somente é realocado se a quantidade de
  agentes que serão inseridos é maior que a quantidade de agentes mortos.
  Antes da inserção o vetor de agentes é particionado, movendo os agentes
  mortos para o início do vetor, facilitando desta forma a inserção dos novos
  agentes. For fim são atualizados os índices para os humanos, pois as
  quantidades de agentes nas localidades foram alterados.

  O método "for_each_n" é responsável pela aplicação do operador
  "InsercaoHumanos" para a inserção dos novos agentes humanos.
*/
void Simulacao::insercaoHumanos() {
  // auto start = high_resolution_clock::now();
  int n = transform_reduce(
            seeds->idx, seeds->idx + 1,
            PreInsercaoHumanos(parametros, ciclo, ambiente),
            0, plus<int>()
          );
  if (n > 0) {
    int m = count_if(
              humanos->humanosDev->begin(),
              humanos->humanosDev->end(),
              EstaMortoHumano()
            );

    if (n > m) {
      humanos->nHumanos += (n - m);
      humanos->humanosDev->resize(humanos->nHumanos, Humano());
      humanos->PhumanosDev = raw_pointer_cast(humanos->humanosDev->data());
    }

    partition(
      humanos->humanosDev->begin(),
      humanos->humanosDev->end(),
      EstaMortoHumano()
    );

    for_each_n(
      seeds->idx, 1,
      InsercaoHumanos(humanos, ambiente, parametros, ciclo, seeds)
    );

    humanos->atualizacaoIndices();
  }
  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<microseconds>(stop - start);
  // cout << "insercaoHumanos(): ";
  // cout << duration.count() << " microsecs" << endl;
}

/*
  Método responsável pelo processamento das saídas resultantes do ciclo de
  simulação. As saídas populacionais são geradas paralelamente para cada
  subpopulação computada. Já as saídas espaciais são geradas paralelamente para
  cada posição do ambiente. As chamadas aos métodos "for_each_n" são responsáveis
  pela aplicação dos operadores sobre os dados.
*/
void Simulacao::computarSaidas() {
  for_each_n(
    seeds->idx, N_COLS_H,
    ContPopTH(humanos, saidas, ciclo - 1)
  );
  for_each_n(
    seeds->idx, N_COLS_H,
    ContPopQH(humanos, saidas, ciclo - 1)
  );
  for_each_n(
    seeds->idx, N_COLS_H,
    ContPopNovoQH(humanos, saidas, ciclo - 1)
  );
  for_each_n(
    seeds->idx, ambiente->sizePos,
    ContEspacialH(
      humanos, saidas, ambiente, parametros->nCiclos, ciclo - 1
    )
  );
  for_each_n(
    seeds->idx, ambiente->sizePos,
    ContEspacialNovoH(
      humanos, saidas, ambiente, parametros->nCiclos, ciclo - 1
    )
  );
  for_each_n(
    seeds->idx, N_COLS_H,
    ContPopNovoTH(humanos, saidas, ciclo - 1)
  );
  for_each_n(
    seeds->idx, N_COLS_H,
    ContPopNovoQH(humanos, saidas, ciclo - 1)
  );

  for_each_n(
    seeds->idx, N_COLS_MD,
    ContPopTMD(mosquitos, saidas, ciclo - 1)
  );
  for_each_n(
    seeds->idx, N_COLS_MD,
    ContPopQMD(mosquitos, saidas, ciclo - 1)
  );
  for_each_n(
    seeds->idx, ambiente->sizePos,
    ContEspacialMD(
      mosquitos, saidas, ambiente, parametros->nCiclos, ciclo - 1
    )
  );
  for_each_n(
    seeds->idx, N_COLS_MD,
    ContPopNovoTMD(mosquitos, saidas, ciclo - 1)
  );

  for_each_n(
    seeds->idx, N_COLS_MW,
    ContPopTMW(mosquitos, saidas, ciclo - 1)
  );
  for_each_n(
    seeds->idx, N_COLS_MW,
    ContPopQMW(mosquitos, saidas, ciclo - 1)
  );
}

/*
  Método responsável pela exibição em tela do consumo de memória total em GPU
  para todas as estruturas de dados presentes na simulação. São utilizados os
  métodos "getMemoriaGPU" das distintas classes com dados relevantes à simulação.
  Como os métodos retornam a quantidade de mémoria em bytes, este valor é
  convertido para MB para facilitar a leitura. São considerados os dados das
  classes "Seeds", "Humanos", "Mosquitos", "Saidas", "Parametros" e "Ambiente".
*/
void Simulacao::exibirConsumoMemoria() {
  double totMem = 0;
  totMem += seeds->getMemoriaGPU();
  totMem += humanos->getMemoriaGPU();
  totMem += mosquitos->getMemoriaGPU();
  totMem += saidas->getMemoriaGPU();
  totMem += parametros->getMemoriaGPU();
  totMem += ambiente->getMemoriaGPU();
  cout << (totMem / (1 << 20)) << "MB" << endl;
}

/*
   Método responsável por gerar as saídas de bitstring
   para agentes do tipo mosquito.
*/
void Simulacao::saidaBitstringMosquitos() {
  // Calcula o total de mosquitos e copia os bitstrings da GPU para a CPU
  int nMosquitos = mosquitos->nMosquitos;
  Mosquito *vetorMosquitos = new Mosquito[nMosquitos]();
  copy_n(mosquitos->mosquitosDev->begin(), nMosquitos, vetorMosquitos);

  string nomeArquivo = string("mosquitos_");
  nomeArquivo += to_string(ciclo - 1);
  if (saidaSubciclo) {
    nomeArquivo += string("-");
    nomeArquivo += to_string(periodo);
    nomeArquivo += string("-");
    nomeArquivo += to_string(subciclo);
  }
  nomeArquivo += string(".csv");

  // Abre o arquivo de saída para mosquitos
  string saida = saidaM + nomeArquivo;
  ofstream arquivo(saida);
  if (not arquivo.is_open()) {
    cerr << "Arquivo: ";
    cerr << saida;
    cerr << " nao foi aberto!" << endl;
    exit(1);
  }

  // cout << "Salvando arquivo " << saida << endl;

  // Escreve os valores dos bitstrings no arquivo de saída
  arquivo << "ID;T1;T2;T3;T4" << endl;
  for (int i = 0; i < nMosquitos; ++i) {
    arquivo << vetorMosquitos[i].id << ";";
    arquivo << vetorMosquitos[i].t1 << ";";
    arquivo << vetorMosquitos[i].t2 << ";";
    arquivo << vetorMosquitos[i].t3 << ";";
    arquivo << vetorMosquitos[i].t4 << endl;
  }
  arquivo.close();

  // Libera a memória alocada para o vetor de mosquitos na CPU
  delete[](vetorMosquitos);
}

/*
   Método responsável por gerar as saídas de bitstring
   para agentes do tipo humano.
*/
void Simulacao::saidaBitstringHumanos() {
  // Calcula o total de humanos e copia os bitstrings da GPU para a CPU
  int nHumanos = humanos->nHumanos;
  Humano *vetorHumanos = new Humano[nHumanos]();
  copy_n(humanos->humanosDev->begin(), nHumanos, vetorHumanos);

  string nomeArquivo = string("humanos_");
  nomeArquivo += to_string(ciclo - 1);
  if (saidaSubciclo) {
    nomeArquivo += string("-");
    nomeArquivo += to_string(periodo);
  }
  nomeArquivo += string(".csv");

  // Abre o arquivo de saída para humanos
  string saida = saidaH + nomeArquivo;
  ofstream arquivo(saida);
  if (not arquivo.is_open()) {
    cerr << "Arquivo: ";
    cerr << saida;
    cerr << " nao foi aberto!" << endl;
    exit(1);
  }

  // cout << "Salvando arquivo " << saida << endl;

  // Escreve os valores dos bitstrings no arquivo de saída
  arquivo << "ID;T1;T2;T3;T4" << endl;
  for (int i = 0; i < nHumanos; ++i) {
    arquivo << vetorHumanos[i].id << ";";
    arquivo << vetorHumanos[i].t1 << ";";
    arquivo << vetorHumanos[i].t2 << ";";
    arquivo << vetorHumanos[i].t3 << ";";
    arquivo << vetorHumanos[i].t4 << endl;
  }
  arquivo.close();

  // Libera a memória alocada para o vetor de humanos na CPU.
  delete[](vetorHumanos);
}

/*
  Método responsável por gerar as saídas de distribuição
  espacial da densidade de ovos.
*/
void Simulacao::saidaOviposicao() {
  int nFocos = ambiente->sizeFocos;
  int *capFocos = new int[nFocos]();
  copy_n(ambiente->capFocosDev->begin(), ambiente->sizeFocos, capFocos);

  ofstream arquivo(arquivoSaidaOviposicao, ofstream::app);
  if (not arquivo.is_open()) {
    cerr << "Arquivo: ";
    cerr << arquivoSaidaOviposicao;
    cerr << " nao foi aberto!" << endl;
    exit(1);
  }

  // cout << "Escrevendo no arquivo " << arquivoSaidaOviposicao << endl;

  arquivo << (ciclo - 1);
  for (int i = 0; i < ambiente->sizeFocos; i++) {
    arquivo << ";" << capFocos[i];
  }
  arquivo << endl;
  arquivo.close();

  delete[](capFocos);
}
