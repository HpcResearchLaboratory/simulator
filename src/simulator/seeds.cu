#include "hip/hip_runtime.h"
#include <simulator/macros/parameters.hpp>
#include <simulator/seeds.hpp>
#include <simulator/util/rand_perc.hpp>

/*
  Operador () da classe InitSeeds.

  Este método é responsável por inicializar as seeds para geração de
  números aleatórios em GPU. O primeiro elemento da tupla "t" é um id númerico
  da seed e o segundo elemento é a estrutura utilizada para geração dos
  números aleatórios.
*/
__host__ __device__ void InitSeeds::operator()(thrust::tuple<int, dre &> t) {
  int seed = get<0>(t);
  get<1>(t) = dre(seed);
}

/*
  Construtor da classe Seeds, que armazena todas as seeds utilizadas para a
  geração de números  aleatórios em GPU durante a execução de simulações.

  A variável "idx" armazena uma instância da classe "counting_iterator", que
  mimetiza um contador infinito, começando em "0". Esta variável é utilizada à
  geração de ids à execução dos métodos em GPU. Cada id indexa um dado que
  pode ser processado paralelamente aos outros dados do conjunto.

  O método realiza a geração de números aleatórios em CPU e os utiliza à
  geração das seeds em GPU.

  A quantidade de seeds geradas é obtida a partir do valor máximo encontrado
  na lista "l". A lista "l" é inicializada com os valores das quantidades de
  agentes humanos, agentes mosquitos e posições do ambiente.
*/
Seeds::Seeds(initializer_list<int> l) {
  // Inicialização do contador em 0.
  idx = make_counting_iterator(0);

  // Obtenção da quantidade de seeds necessárias.
  nSeeds = max(l);

  RandPerc rand;

  // Geração de números aleatórios em CPU.
  seedsDev = new DVector<dre>(nSeeds);
  int *rands = new int[nSeeds]();
  generate(rands, rands + nSeeds,
           [&]() { return (int)ENTRE_FAIXA(0, 100000, rand()); });
  DVector<int> randsDev(rands, rands + nSeeds);
  delete[] (rands);

  // Inicialização das seeds em GPU.
  for_each_n(make_zip_iterator(make_tuple(randsDev.begin(), seedsDev->begin())),
             nSeeds, InitSeeds());
  PseedsDev = raw_pointer_cast(seedsDev->data());
}

/*
  Método responsável pela obtenção do consumo de memória da classe Seeds.
*/
int Seeds::getMemoriaGPU() { return (nSeeds * sizeof(dre)); }

/*
  Destrutor da classe Seeds
*/
Seeds::~Seeds() { delete (seedsDev); }
