#include "hip/hip_runtime.h"
#include "Contato.h"
#include "Fontes/Ambiente.h"
#include "Fontes/Parametros.h"
#include "Fontes/Seeds.h"
#include "Fontes/Mosquitos/Mosquitos.h"
#include "Fontes/Macros/MacrosMosquitos.h"
#include "Fontes/Macros/2_CON_M.h"
#include "Fontes/Macros/MacrosGerais.h"

/*
  Este operador é paralelizado para cada POSIÇÃO do ambiente. 

  Este método é responsável pela aplicação do operador de contato entre agentes
  mosquitos, que representa o acasalamento entre os agentes machos e fêmeas.

  Inicialmente, são filtradas todas as fêmeas na fase alada que ainda não
  acasalaram.

  Para cada fêmea, determina-se o melhor candidato para reprodução que se
  encontra na mesma posição da fêmea, por meio da função getMacho(). Caso
  nenhum macho seja encontrado, a fêmea em questão não acasala.

  Finalmente, determina-se as características da prole de acordo com os
  atributos da fêmea e do macho, considerando a infecção por Wolbachia.
*/
ContatoMosquitos::ContatoMosquitos(
  Mosquitos *mosquitos, Ambiente *ambiente, Parametros *parametros,
  int periodo, Seeds *seeds
) {
  this->mosquitos = mosquitos->PmosquitosDev;
  this->indMosquitos = mosquitos->PindMosquitosDev;
  this->parametros = parametros->PparametrosDev;
  this->pos = ambiente->PposDev;
  this->periodo = periodo;
  this->seeds = seeds->PseedsDev;
}

/*
  Operador () da classe ContatoMosquitos.
*/
__host__ __device__
void ContatoMosquitos::operator()(int id) {
  dre& seed = seeds[id];
  urd<double> dist(0.0, 1.0);

  int x = pos[id].x, y = pos[id].y;
  int l = pos[id].lote, q = pos[id].quadra;

  // Índices correspondentes à quadra atual para o vetor de mosquitos
  int indInicial = indMosquitos[q];
  int indFinal = indMosquitos[q + 1];

  // Percorre todos os mosquitos que estão nesta posição, filtrando fêmeas na
  // fase alada e ainda não acasaladas
  for (int idFemea = indInicial; idFemea < indFinal; ++idFemea) {

    // Acesso aos atributos do mosquito fêmea
    int q_f  = GET_Q_M(idFemea);
    int l_f = GET_L_M(idFemea);
    int x_f  = GET_X_M(idFemea);
    int y_f = GET_Y_M(idFemea);
    int sw_f = GET_SW_M(idFemea);
    int s_f = GET_S_M(idFemea);
    int vd_f = GET_VD_M(idFemea);
    int fs_f = GET_FS_M(idFemea);
    int ta_f = GET_TA_M(idFemea);

    // Ignora agentes fora desta posição
    if (l_f != l or x_f != x or y_f != y) continue;

    // Ignora agentes mortos ou machos
    if (vd_f == MORTO or s_f != FEMEA) continue;

    // Ignora ovos, larvas e pupas
    if (fs_f != ATIVA and fs_f != DECADENTE) continue;

    // Ignora fêmeas acasaladas
    if (ta_f != NENHUM) continue;

    // A fêmea procura um macho para acasalamento. 
    int idMacho = getMacho(x_f, y_f, l_f, q_f);
    if (idMacho == -1) continue;

    // Acesso aos atributos do mosquito macho
    int c_m = GET_C_M(idMacho);
    int sw_m = GET_SW_M(idMacho);

    int tipoAcasal, tipoProle;
    double fatorPref, probAcasal;

    // Tipo de acasalamento, prole, fator de preferência e probabilidade de
    // acasalamento são definidos de acordo com os estados da fêmea e do macho
    switch (sw_m) {
      case SAUDAVEL: {
        switch (sw_f) {
          case SAUDAVEL: { // Fêmea e macho saudáveis
            tipoAcasal = ACA_SAUDAVEL;
            tipoProle = SAUDAVEL;
            fatorPref = pow(K1, c_m);
            probAcasal = (c_m == 0) ? BETASS22 : BETASS23;
          } break;
          case WOLBACHIA: { // Fêmea infectada com Wolbachia e macho saudável
            tipoAcasal = ACA_SAUDAVEL;
            tipoProle = WOLBACHIA;
            fatorPref = pow(K1, c_m);
            probAcasal = (c_m == 0) ? BETAIS22 : BETAIS23;
          } break;
        }
      } break;
      case WOLBACHIA: {
        switch (sw_f) {
          case SAUDAVEL: { // Fêmea saudável e macho infectado com Wolbachia
            tipoAcasal = ACA_INFECTADO;
            tipoProle = ESTERIL;
            fatorPref = pow(K2, c_m);
            probAcasal = (c_m == 0) ? BETASI22 : BETASI23;
          } break;
          case WOLBACHIA: { // Fêmea e macho infectados com Wolbachia
            tipoAcasal = ACA_INFECTADO;
            tipoProle = WOLBACHIA;
            fatorPref = pow(K2, c_m);
            probAcasal = (c_m == 0) ? BETAII22 : BETAII23;
          } break;
        }
      } break;
    }

    double taxaSucesso = (fatorPref * probAcasal * TAXA_FECUNDIDADE_FEMEA);

    // Aplica a redução de atividade noturna
    if (periodo == NOITE) taxaSucesso *= REDUCAO_CONTATO_M;

    if (randPerc <= taxaSucesso) {
      // Altera o tipo de acasalamento e prole da fêmea
      SET_TA_M(idFemea, tipoAcasal);
      SET_PR_M(idFemea, tipoProle);

      // Zera os contadores de latência e de posturas da fêmea
      SET_C_M(idFemea, 0);
      SET_CP_M(idFemea, 0);

      // Incrementa o contador de acasalamentos do macho
      SET_C_M(idMacho, c_m + 1);
    }
  }
}

/*
  Este método encontra o melhor candidato para reprodução na posição designada.
  Para isso, retorna o id do mosquito macho com o menor número de acasalamentos
  que for encontrado. Caso não exista nenhum agente compatível, retorna -1.
*/
__host__ __device__
int ContatoMosquitos::getMacho(int x, int y, int l, int q) {
  int id = -1;
  int nAcasal = INT_MAX;

  // Índices correspondentes à quadra atual para o vetor de mosquitos
  int indInicial = indMosquitos[q];
  int indFinal = indMosquitos[q + 1];

  for (int idMacho = indInicial; idMacho < indFinal; ++idMacho) {
    // Acesso aos atributos do agente
    int l_m = GET_L_M(idMacho);
    int x_m  = GET_X_M(idMacho);
    int y_m = GET_Y_M(idMacho);
    int s_m  = GET_S_M(idMacho);
    int c_m = GET_C_M(idMacho);
    int vd_m = GET_VD_M(idMacho);
    int fs_m = GET_FS_M(idMacho);

    // Ignora agentes fora desta posição
    if (l_m != l or x_m != x or y_m != y) continue;

    // Ignora agentes mortos ou fêmeas
    if (vd_m == MORTO or s_m != MACHO) continue;

    // Ignora ovos, larvas e pupas
    if (fs_m != ATIVA and fs_m != DECADENTE) continue;

    if (c_m < nAcasal) {
      id = idMacho;
      nAcasal = c_m;
    }
  }
  return id;
}
