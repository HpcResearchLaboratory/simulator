#include "hip/hip_runtime.h"
#include "Geracao.h"
#include "Fontes/Ambiente.h"
#include "Fontes/Parametros.h"
#include "Fontes/Macros/1_MOV_M.h"
#include "Fontes/Macros/5_GER_M.h"
#include "Fontes/Mosquitos/Mosquitos.h"
#include "Fontes/Macros/MacrosMosquitos.h"
#include "Fontes/Macros/MacrosGerais.h"
#include "Fontes/Seeds.h"

/*
  Este operador é paralelizado para cada AGENTE.
  
  Este operador é responsável pela estimativa da quantidade de ovos que serão
  inseridos no ambiente em decorrência do acasalamento entre mosquitos.

  Para isso, inicialmente são filtradas todas as fêmeas acasaladas.
  Para cada fêmea, são analisados os seguintes atributos:
    CE - Contador de ciclos entre posturas
    CG - Ciclos de gestação

  Antes de estar apta para postura, cada fêmea deve esperar um certo número
  de ciclos, representado pelos parâmetros:
    INTERVALO_ENTRE_POSTURAS_FEMEA + CICLOS_GESTACAO

  Além disso, para a postura de ovos a fêmea deve estar alimentada (repasto)
  e deve estar em um ponto de foco.

  Caso as condições para postura estejam cumpridas, a fêmea é marcada como
  apta para postura (FG - Flag de geração de ovos), e seus contadores são
  reiniciados.

  Caso a fêmea seja estéril ou esteja fora de uma posição de foco, a postura
  será realizada entretanto nenhum ovo sobreviverá, portanto seus ovos não são
  contabilizados.
*/
PreGeracao::PreGeracao(
  Ambiente *ambiente, Mosquitos *mosquitos,
  Parametros *parametros, Seeds *seeds
) {
  this->mosquitos = mosquitos->PmosquitosDev;
  this->parametros = parametros->PparametrosDev;
  this->seeds = seeds->PseedsDev;
  this->indFocos = ambiente->PindFocosDev;
  this->focos = ambiente->PfocosDev;
  this->pos = ambiente->PposDev;
  this->indQuadras = ambiente->PindQuadrasDev;
}

/*
  Operador () da classe PreGeracao.
*/
__host__ __device__
int PreGeracao::operator()(int id) {
  dre& seed = seeds[id];
  urd<double> dist(0.0, 1.0);
  int nMosquitos = 0;

  int vd_m = GET_VD_M(id);
  int s_m = GET_S_M(id);
  int fs_m = GET_FS_M(id);
  int ta_m = GET_TA_M(id);

  // Filtra apenas fêmeas acasaladas
  if (vd_m == MORTO or s_m == MACHO) return 0;
  if (fs_m != ATIVA and fs_m != DECADENTE) return 0;
  if (ta_m == NENHUM) return 0;

  int ce_m = GET_CE_M(id);
  int cg_m = GET_CG_M(id);
  int pr_m = GET_PR_M(id);
  int am_m = GET_AM_M(id);

  if (ce_m < INTERVALO_ENTRE_POSTURAS_FEMEA) {
    SET_CE_M(id, ce_m + 1);
  } else { // Caso o intervalo entre posturas tenha acabado
    if (cg_m < CICLOS_GESTACAO) {
      SET_CG_M(id, cg_m + 1);
    } else { // Caso a gestação esteja concluída
      // Reinicia os contadores
      SET_CE_M(id, 0);
      SET_CG_M(id, 0);
      // Contabiliza os ovos e marca a fêmea como apta, caso esteja
      if (am_m == ALIMENTADO) {
        // Ativa a flag de geração de ovos
        SET_FG_M(id, 1);
        // Ignora fêmeas estéreis e fora de posições de foco
        if (pr_m != ESTERIL and estaEmFoco(id)) {
          nMosquitos += max(AS21_MAX, AI21_MAX);
        }
      }
    }
  }

  return nMosquitos;
}

/*
  Verifica se a posição atual de determinado mosquito é um ponto de foco. 
*/
__host__ __device__
bool PreGeracao::estaEmFoco(int id) {
  int x = GET_X_M(id); int y = GET_Y_M(id);
  int q = GET_Q_M(id); int l = GET_L_M(id);

  int inicioFocos = indFocos[indQuadras[q * 2] + l];
  int finalFocos = indFocos[indQuadras[q * 2] + l + 1];

  if (inicioFocos >= finalFocos) return false;
  
  for (int i = inicioFocos; i < finalFocos; i++) {
    if (x == pos[focos[i]].x and y == pos[focos[i]].y) {
      return true;
    }
  }
  return false;
}

/*
  Este operador é paralelizado para cada AGENTE.

  Este operador determina e insere os ovos que entrarão no ambiente resultantes
  da postura das fêmeas do mosquito.

  Inicialmente, são filtradas apenas as fêmeas acasaladas e marcadas como
  aptas para postura.

  O contador de postura é incrementado, e a partir deste valor verifica-se
  quando a fêmea necessita realizar repasto novamente. Nesse caso, a flag de
  alimentação do agente é desativada.

  Para fêmeas estéreis ou que não se encontram em um ponto de foco, os ovos
  são descartados (não sobrevivem). Além disso, caso a realocação do vetor não
  tenha ocorrido por limitações de memória, a postura também é cancelada.

  Finalmente, para as fêmeas que cumprem os requisitos, determina-se a
  quantidade e o tipo de ovos para inserção no ambiente de simulação.
*/
Geracao::Geracao(
  Ambiente *ambiente, Mosquitos *mosquitos,
  Parametros *parametros, Seeds *seeds
) {
  this->mosquitos = mosquitos->PmosquitosDev;
  this->nMosquitos = mosquitos->nMosquitos;
  this->parametros = parametros->PparametrosDev;
  this->alocarMosquitos = mosquitos->alocarMosquitos;
  this->seeds = seeds->PseedsDev;
  this->indFocos = ambiente->PindFocosDev;
  this->focos = ambiente->PfocosDev;
  this->pos = ambiente->PposDev;
  this->indQuadras = ambiente->PindQuadrasDev;
  this->capFocos = ambiente->PcapFocosDev;
}

/*
  Operador () da classe Geracao.
*/
__host__ __device__
void Geracao::operator()(int id) {
  // dre& seed = seeds[id];
  urd<double> dist(0.0, 1.0);
  int i = 0;

  for (int id = 0; id < nMosquitos; ++id) {

    int vd_m = GET_VD_M(id);
    int s_m = GET_S_M(id);
    int fs_m = GET_FS_M(id);
    int ta_m = GET_TA_M(id);
    int fg_m = GET_FG_M(id);
    int cp_m = GET_CP_M(id);
    int pr_m = GET_PR_M(id);

    // Filtra apenas fêmeas acasaladas
    if (vd_m == MORTO or s_m == MACHO) continue;
    if (fs_m != ATIVA and fs_m != DECADENTE) continue;
    if (ta_m == NENHUM) continue;

    // Filtra fêmeas aptas para postura
    if (fg_m == 0) continue;

    // Incrementa o contador de posturas da fêmea e desativa a flag de geração
    SET_CP_M(id, cp_m + 1);
    SET_FG_M(id, 0);

    // Marca a fêmea como não alimentada
    SET_AM_M(id, NAO_ALIMENTADO);

    // Ignora fêmeas estéreis ou fora de posições de foco
    if (pr_m == ESTERIL or not estaEmFoco(id)) continue;

    // A postura não ocorre caso a realocação do vetor não tenha ocorrido
    if (alocarMosquitos) postura(id, i);
  }
}

/*
  Este método determina a quantidade e tipo dos ovos a serem
  inseridos a partir dos atributos internos da fêmea.
*/
__host__ __device__
void Geracao::postura(int id, int &i) {
  dre& seed = seeds[id];
  urd<double> dist(0.0, 1.0);

  int pr_m = GET_PR_M(id);
  int fs_m = GET_FS_M(id);

  switch (pr_m) {
    case SAUDAVEL: {
      switch (fs_m) {
        case ATIVA: {
          inserirOvos(id, i, AS21, PS21, SAUDAVEL);
        } break;
        case DECADENTE: {
          inserirOvos(id, i, H1 * AS21, PS31, SAUDAVEL);
        } break;
      }
    } break;
    case WOLBACHIA: {
      switch (fs_m) {
        case ATIVA: {
          inserirOvos(id, i, AI21, PI21, WOLBACHIA);
        } break;
        case DECADENTE: {
          inserirOvos(id, i, H2 * AI21, PI31, WOLBACHIA);
        } break;
      }
    } break;
  }
}

/*
  Organiza e executa a inserção de uma determinada
  quantidade de ovos com os atributos especificados.
*/
__host__ __device__
void Geracao::inserirOvos(int id, int& i, int total, double frac, int sw) {
  dre& seed = seeds[id];
  urd<double> dist(0.0, 1.0);
  int x = GET_X_M(id), y = GET_Y_M(id);
  int l = GET_L_M(id), q = GET_Q_M(id);

  // Calcula o número de ovos que podem efetivamente ser inseridos,
  // considerando o limite dos focos.
  int idFoco = getIdFoco(id);
  if (idFoco == -1) return;
  int espacoParaOvos = LIMITE_FOCOS - capFocos[idFoco];
  if (espacoParaOvos <= 0) return;
  
  // Determina o número de machos e fêmeas
  int inseridos = (total > espacoParaOvos) ? espacoParaOvos : total;
  int nFemeas = lround(inseridos * frac);
  int nMachos = inseridos - nFemeas;

  // Contabiliza os ovos inseridos no foco
  capFocos[idFoco] += inseridos;

  // Inserção dos ovos machos.
  inserirMosquitos(nMachos, MACHO, sw, OVO, SUSCETIVEL, 0, q, l, x, y, i);
  // Inserção dos ovos fêmeas.
  inserirMosquitos(nFemeas, FEMEA, sw, OVO, SUSCETIVEL, 0, q, l, x, y, i);
}

/*
  Organiza e executa a inserção de uma determinada
  quantidade de mosquitos com os atributos especificados.
*/
__host__ __device__
void Geracao::inserirMosquitos(
  int quantidade, int s, int sw, int fs, int sd, int st,
  int q, int l, int x, int y, int& i
) {
  for (int j = 0; j < quantidade; ++j) {
    // Inicialização do novo agente.
    inicializarMosquito(i++, s, sw, fs, sd, st, q, l, x, y);
  }
}

/*
  Método de inicialização de agentes mosquitos.
  São atribuídos valores padrão para os atributos necessários,
  além de valores determinados pelos parâmetros desta função.
*/
__host__ __device__
void Geracao::inicializarMosquito(
  int id, int s, int sw, int fs, int sd, int st,
  int q, int l, int x, int y
) {
  SET_S_M(id, s);
  SET_SW_M(id, sw);
  SET_FS_M(id, fs);
  SET_IE_M(id, 0);
  SET_SD_M(id, sd);
  SET_ST_M(id, st);
  SET_VD_M(id, VIVO);
  SET_C_M(id, 0);

  SET_CR_M(id, 0);
  SET_FG_M(id, 0);
  SET_TI_M(id, SEM_INFLUENCIA);
  SET_FM_M(id, 1);
  SET_FP_M(id, 0);
  SET_FV_M(id, 0);
  SET_CG_M(id, 0);
  SET_CE_M(id, 0);
  SET_PR_M(id, NENHUM);
  SET_AM_M(id, 0);
  SET_TA_M(id, NENHUM);
  SET_CP_M(id, 0);

  SET_X_M(id, x);
  SET_Q_M(id, q);

  SET_Y_M(id, y);
  SET_L_M(id, l);

  // Sorteia um identificador para o novo agente
  dre& seed = seeds[id];
  urd<double> dist(0.0, 1.0);
  SET_ID_M(id, ENTRE_FAIXA(0, MAX_UINT32, randPerc));
}

/*
  Verifica se a posição atual de determinado mosquito é um ponto de foco. 
*/
__host__ __device__
bool Geracao::estaEmFoco(int id) {
  int x = GET_X_M(id); int y = GET_Y_M(id);
  int q = GET_Q_M(id); int l = GET_L_M(id);

  int inicioFocos = indFocos[indQuadras[q * 2] + l];
  int finalFocos = indFocos[indQuadras[q * 2] + l + 1];

  if (inicioFocos >= finalFocos) return false;
  
  for (int i = inicioFocos; i < finalFocos; i++) {
    if (x == pos[focos[i]].x and y == pos[focos[i]].y) {
      return true;
    }
  }
  return false;
}

/*
  Caso o agente esteja em um ponto de foco,
  retorna o índice desta posição no vetor de focos.
*/
__host__ __device__
int Geracao::getIdFoco(int id) {
  int x = GET_X_M(id); int y = GET_Y_M(id);
  int q = GET_Q_M(id); int l = GET_L_M(id);

  int inicioFocos = indFocos[indQuadras[q * 2] + l];
  int finalFocos = indFocos[indQuadras[q * 2] + l + 1];

  if (inicioFocos >= finalFocos) return -1;
  
  for (int i = inicioFocos; i < finalFocos; i++) {
    if (x == pos[focos[i]].x and y == pos[focos[i]].y) {
      return i;
    }
  }
  return -1;
}
