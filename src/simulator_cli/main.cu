#include <simulator/monte_carlo.hpp>
#include <simulator/parameters.hpp>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <cstddef>
#include <cstdlib>
#include <iostream>

#include <cxxopts.hpp>

auto main(int argc, char** argv) -> int {
  cxxopts::Options options("simulator",
                           "Command line disease spread simulator");
  // clang-format off
  options.add_options()
    ("d,device", "CUDA device to run", cxxopts::value<int>()->default_value("0"))
    ("n,number_of_mc_simulations", "Number of simulations to run in the monte carlo strategy", cxxopts::value<int>()->default_value("1"))
    ("s,subcycle_output", "Wheather to print or not data in every subcycle", cxxopts::value<bool>()->default_value("false"))
    ("h,help", "Print help")
  ;
  // clang-format on

  auto result = options.parse(argc, argv);
  if (result.count("help")) {
    std::cout << options.help() << std::endl;
    std::exit(0);
  }

  // auto device = result["device"].as<int>();
  // auto number_of_simulations = result["number_of_mc_simulations"].as<int>();
  // auto subcycle_output = result["subcycle_output"].as<bool>();

  auto parameters = simulator::Parameters::from_dir("assets/input/parameters");

  // hipSetDevice(device);
  // for (int i = 0; i < number_of_simulations; ++i)
  //   simulator::MonteCarlo { "assets/input/mc" + std::to_string(i),
  //                           "assets/output/mc" + std::to_string(i),
  //                           subcycle_output }
  //     .run();
}
